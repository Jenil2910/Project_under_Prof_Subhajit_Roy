#include "hip/hip_runtime.h"
#include<iostream>
#include<nvgraph.h>
#include<bits/stdc++.h>
#include<stdio.h>

#define ROOT_TWO sqrt(2)
using namespace std;
///////////////////////////////////////////////////
void check(nvgraphStatus_t status) {
  if (status != NVGRAPH_STATUS_SUCCESS) {
    printf("ERROR : %d\n", status);
    exit(0);
  }
}

float* time_finder(float **slow,int _N,int _M) {
    int edges=_N *( _M +1)+(_N+1)*_M+ _M * _N*2;
    int nodes=(_N+1)*(_M+1);
    float* weights=new float[edges*2];
    int* destination_offset=new int[nodes+1];
    int* source=new int[edges*2];
    int count=0;
    for(int i=0;i<= _N;i++){
        for(int j=0;j<= _M;j++){
            int index=i*(_M+1)+j;
            destination_offset[index]=count;
            //case 1
            if(i>0){
                //case 1.1
                if(j>0){
                    weights[count]= ROOT_TWO * slow[i-1][j-1];
                    source[count]=(i-1)*(_M+1)+j-1;
                    count++;
                }
                //case 1.2
                if(j>0&&j<_M){
                    weights[count]=(slow[i-1][j-1]+slow[i-1][j])/2.0;
                    source[count]=(i-1)*(_M+1)+j;
                    count++;
                }else if(j==_M){
                    weights[count]=slow[i-1][j-1];
                    source[count]=(i-1)*(_M+1)+j;
                    count++;
                }else{//j==0
                    weights[count]=slow[i-1][j];
                    source[count]=(i-1)*(_M+1)+j;
                    count++;
                }
                //case 1.3
                if(j<_M){
                    weights[count]= ROOT_TWO * slow[i-1][j];
                    source[count]=(i-1)*(_M+1)+j+1;
                    count++;
                }
            }
            //case 2
            //case 2.1
            if(i>0&&j>0&&i<_N){
                weights[count]=(slow[i-1][j-1]+slow[i][j-1])/2.0;
                source[count]=(i)*(_M+1)+j-1;
                count++;
            }else if(i==0&&j>0){
                weights[count]=slow[i][j-1];
                source[count]=(i)*(_M+1)+j-1;
                count++;
            }else if(i==_N&&j>0){
                weights[count]=slow[i-1][j-1];
                source[count]=(i)*(_M+1)+j-1;
                count++;
            }
            //case 2.2
            if(i>0&&j<_M&&i<_N){
                weights[count]=(slow[i-1][j]+slow[i][j])/2.0;
                source[count]=(i)*(_M+1)+j+1;
                count++;
            }else if(i==0&&j<_M){
                weights[count]=slow[i][j];
                source[count]=(i)*(_M+1)+j+1;
                count++;
            }else if(i==_N&&j<_M){
                weights[count]=slow[i-1][j];
                source[count]=(i)*(_M+1)+j+1;
                count++;
            }
            //case 3
            if(i<_N){
                //case 3.1
                if(j>0){
                    weights[count]= ROOT_TWO * slow[i][j-1];
                    source[count]=(i+1)*(_M+1)+j-1;
                    count++;
                }
                //case 3.2
                if(j>0&&j<_M){
                    weights[count]=(slow[i][j-1]+slow[i][j])/2.0;
                    source[count]=(i+1)*(_M+1)+j;
                    count++;
                }else if(j==_M){
                    weights[count]=slow[i][j-1];
                    source[count]=(i+1)*(_M+1)+j;
                    count++;
                }else{//j==0
                    weights[count]=slow[i][j];
                    source[count]=(i+1)*(_M+1)+j;
                    count++;
                }
                //case 3.3
                if(j<_M){
                    weights[count]= ROOT_TWO * slow[i][j];
                    source[count]=(i+1)*(_M+1)+j+1;
                    count++;
                }
            }
        }
    }
    destination_offset[nodes]=count;
	//Converting Adjacency Matrix in input to required input for nvgraph


    const size_t n = nodes, nnz = edges*2,    vertex_numsets = 1,    edge_numsets = 1;
    float weights_h[nnz];
    int destination_offsets_h[nodes+1];
    int source_indices_h[nnz];

    for(int i=0;i<nnz;i++){
    	weights_h[i]=weights[i];
    	source_indices_h[i]=source[i];
    }
    for(int i=0;i<nodes+1;i++){
    destination_offsets_h[i]=destination_offset[i];
    }

    //Converting our variables to variables for nvgraph

    float * sssp_1_h;
    void * * vertex_dim; // nvgraph variables _h for host data.
    nvgraphStatus_t status;
    nvgraphHandle_t handle;
    nvgraphGraphDescr_t graph;
    nvgraphCSCTopology32I_t CSC_input;
    hipDataType edge_dimT = HIP_R_32F;
    hipDataType * vertex_dimT;


    // Init host data
    sssp_1_h = (float * ) malloc(n * sizeof(float));
    vertex_dim = (void * * ) malloc(vertex_numsets * sizeof(void * ));
    vertex_dimT = (hipDataType * ) malloc(vertex_numsets * sizeof(hipDataType));
    CSC_input = (nvgraphCSCTopology32I_t) malloc(sizeof(struct nvgraphCSCTopology32I_st));
    vertex_dim[0] = (void * ) sssp_1_h;
    vertex_dimT[0] = HIP_R_32F;


    check(nvgraphCreate( & handle));
    check(nvgraphCreateGraphDescr(handle, & graph));
    CSC_input -> nvertices = n;
    CSC_input -> nedges = nnz;
    CSC_input -> destination_offsets = destination_offsets_h;
    CSC_input -> source_indices = source_indices_h;
    check(nvgraphSetGraphStructure(handle, graph, (void * ) CSC_input, NVGRAPH_CSC_32));
    check(nvgraphAllocateVertexData(handle, graph, vertex_numsets, vertex_dimT));
    check(nvgraphAllocateEdgeData(handle, graph, edge_numsets, & edge_dimT));
    check(nvgraphSetEdgeData(handle, graph, (void * ) weights_h, 0));
    int source_vert = 0;
    check(nvgraphSssp(handle, graph, 0, & source_vert, 0));
    check(nvgraphGetVertexData(handle, graph, (void * ) sssp_1_h, 0));


    float* Actual_seed = new float[(_N+1)*(_M+1)];
    double error=0;
    free(vertex_dim);
    free(vertex_dimT);
    free(CSC_input);
    check(nvgraphDestroyGraphDescr(handle, graph));
    check(nvgraphDestroy(handle));
    return sssp_1_h;
}
int main(){
  float ** inp;
  inp=(float**)malloc(sizeof(float*)*10);
  for(int i=0;i<10;i++){
    inp[i]=(float*)malloc(sizeof(float)*20);
    for(int j=0;j<20;j++){
      inp[i][j]=1;
    }
  }
  float* out=time_finder(inp,10,20);
  for (int i = 0; i < 21; i++) {
    cout<<out[i]<<' ';
  }
  return 0;
}
