#include "hip/hip_runtime.h"
#include<nvgraph.h>
#include<bits/stdc++.h>
#define ROOT_TWO sqrt(2)
using namespace std;
///////////////////////////////////////////////////
void check(nvgraphStatus_t status) {
  if (status != NVGRAPH_STATUS_SUCCESS) {
    printf("ERROR : %d\n", status);
    exit(0);
  }
}
#define ROW 500
#define COL 4000
int n=(ROW+1)*(COL+1), vertex_numsets = 1, edge_numsets = 1, nnz=ROW *( COL +1)+(ROW+1)*COL+ COL * ROW*2*2;//nnz=edges*2;
double *weights_h=new double[nnz];
int *destination_offsets_h=new int[n+1];
int *source_indices_h=new int[nnz];

double* time_finder(double **slow,int _N,int _M) {
    int count=0;
    for(int i=0;i<= _N;i++){
        for(int j=0;j<= _M;j++){
            int index=i*(_M+1)+j;
            destination_offsets_h[index]=count;
            int i_m,i_p,j_m,j_p;
            i_m=(i-1>0?i-1:0);
            i_p=(i>_N-1?_N-1:i);
            j_m=(j-1>0?j-1:0);
            j_p=(j>_M-1?_M-1:j);
            //case 1
            if(i>0){
                //case 1.1
                if(j>0){
                    weights_h[count]= ROOT_TWO * slow[i_m][j_m];
                    source_indices_h[count]=(i-1)*(_M+1)+j_m;
                    count++;
                }
                //case 1.2
                    weights_h[count]=(slow[i_m][j_m]+slow[i_m][j_p])/2.0;
                    source_indices_h[count]=(i-1)*(_M+1)+j;
                    count++;
                //case 1.3
                if(j<_M){
                    weights_h[count]= ROOT_TWO * slow[i-1][j];
                    source_indices_h[count]=(i-1)*(_M+1)+j+1;
                    count++;
                }
            }
            //case 2
            //case 2.1
            if(j>0){
                weights_h[count]=(slow[i_m][j_m]+slow[i_p][j_m])/2.0;
                source_indices_h[count]=(i)*(_M+1)+j-1;
                count++;
              }
            //case 2.2
            if(j<_M){
                weights_h[count]=(slow[i_m][j_p]+slow[i_p][j_p])/2.0;
                source_indices_h[count]=(i)*(_M+1)+j+1;
                count++;
              }
            //case 3
            if(i<_N){
                //case 3.1
                if(j>0){
                    weights_h[count]= ROOT_TWO * slow[i][j-1];
                    source_indices_h[count]=(i+1)*(_M+1)+j-1;
                    count++;
                }
                //case 3.2
                    weights_h[count]=(slow[i_p][j_m]+slow[i_p][j_p])/2.0;
                    source_indices_h[count]=(i+1)*(_M+1)+j;
                    count++;
                //case 3.3
                if(j<_M){
                    weights_h[count]= ROOT_TWO * slow[i][j];
                    source_indices_h[count]=(i+1)*(_M+1)+j+1;
                    count++;
                }
            }
        }
    }
    destination_offsets_h[n]=count;
	//Converting Adjacency Matrix in input to required input for nvgraph

    double * sssp_1_h;
    void * * vertex_dim; // nvgraph variables _h for host data.
    nvgraphStatus_t status;
    nvgraphHandle_t handle;
    nvgraphGraphDescr_t graph;
    nvgraphCSCTopology32I_t CSC_input;
    hipDataType edge_dimT = HIP_R_32F;
    hipDataType * vertex_dimT;


    // Init host data
    /* *weights_h, *destination_offsets_h, *source_indices_h, n, nnz, vertex_numsets , edge_numsets already defined */
    sssp_1_h = new double[n];
    vertex_dim = new void*[vertex_numsets];
    vertex_dimT=new hipDataType[vertex_numsets];
    CSC_input= (nvgraphCSCTopology32I_t)new nvgraphCSCTopology32I_t;
    vertex_dim[0] = (void * ) sssp_1_h;
    vertex_dimT[0] = HIP_R_32F;


    check(nvgraphCreate( & handle));
    check(nvgraphCreateGraphDescr(handle, & graph));
    CSC_input -> nvertices = n;
    CSC_input -> nedges = nnz;
    CSC_input -> destination_offsets = destination_offsets_h;
    CSC_input -> source_indices = source_indices_h;
    check(nvgraphSetGraphStructure(handle, graph, (void * ) CSC_input, NVGRAPH_CSC_32));
    check(nvgraphAllocateVertexData(handle, graph, vertex_numsets, vertex_dimT));
    check(nvgraphAllocateEdgeData(handle, graph, edge_numsets, & edge_dimT));
    check(nvgraphSetEdgeData(handle, graph, (void * ) weights_h, 0));
    int source_vert = 0;
    check(nvgraphSssp(handle, graph, 0, & source_vert, 0));
    check(nvgraphGetVertexData(handle, graph, (void * ) sssp_1_h, 0));


    double* Actual_seed = new double[(_N+1)*(_M+1)];
    delete weights_h;
    delete destination_offsets_h;
    delete source_indices_h;
    delete vertex_dim;
    delete vertex_dimT;
    delete CSC_input;
    check(nvgraphDestroyGraphDescr(handle, graph));
    check(nvgraphDestroy(handle));
    return sssp_1_h;
}
int main(){
  double ** inp;
  int n=ROW,m=COL;
  inp=new double*[n];
  for(int i=0;i<n;i++){
    inp[i]=new double[m];
    for(int j=0;j<m;j++){
      inp[i][j]=1;
    }
  }
  double tpck[m+1];
  double* out=time_finder(inp,n,m);
  for(int i=0;i<m+1;i++){
    tpck[i]=2;

  }
  double sum=0;
  for(int i=0;i<m+1;i++){
    cout<<out[i]<<" ";
    sum=sum+(tpck[i]-out[i])*(tpck[i]-out[i]);
  }
  cout<<endl<<sum<<endl;
  return 0;
}
