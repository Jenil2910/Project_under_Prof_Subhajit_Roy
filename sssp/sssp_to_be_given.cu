#include "hip/hip_runtime.h"
#include<iostream>
#include<nvgraph.h>
#include<bits/stdc++.h>
#include<stdio.h>

#define ROOT_TWO sqrt(2)
using namespace std;
///////////////////////////////////////////////////
void check(nvgraphStatus_t status) {
  if (status != NVGRAPH_STATUS_SUCCESS) {
    printf("ERROR : %d\n", status);
    exit(0);
  }
}
#define ROW 500
#define COL 4000
float* time_finder(float **slow,int _N,int _M) {
    int edges=_N *( _M +1)+(_N+1)*_M+ _M * _N*2;
    int nodes=(_N+1)*(_M+1);
    float* weights=new float[edges*2];
    int* destination_offset=new int[nodes+1];
    int* source=new int[edges*2];
    int count=0;
    for(int i=0;i<= _N;i++){
        for(int j=0;j<= _M;j++){
            int index=i*(_M+1)+j;
            destination_offset[index]=count;
            int i_m,i_p,j_m,j_p;
            i_m=(i-1>0?i-1:0);
            i_p=(i>_N-1?_N-1:i);
            j_m=(j-1>0?j-1:0);
            j_p=(j>_M-1?_M-1:j);
            //case 1
            if(i>0){
                //case 1.1
                if(j>0){
                    weights[count]= ROOT_TWO * slow[i_m][j_m];
                    source[count]=(i-1)*(_M+1)+j_m;
                    count++;
                }
                //case 1.2
                    weights[count]=(slow[i_m][j_m]+slow[i_m][j_p])/2.0;
                    source[count]=(i-1)*(_M+1)+j;
                    count++;
                //case 1.3
                if(j<_M){
                    weights[count]= ROOT_TWO * slow[i-1][j];
                    source[count]=(i-1)*(_M+1)+j+1;
                    count++;
                }
            }
            //case 2
            //case 2.1
            if(j>0){
                weights[count]=(slow[i_m][j_m]+slow[i_p][j_m])/2.0;
                source[count]=(i)*(_M+1)+j-1;
                count++;
              }
            //case 2.2
            if(j<_M){
                weights[count]=(slow[i_m][j_p]+slow[i_p][j_p])/2.0;
                source[count]=(i)*(_M+1)+j+1;
                count++;
              }
            //case 3
            if(i<_N){
                //case 3.1
                if(j>0){
                    weights[count]= ROOT_TWO * slow[i][j-1];
                    source[count]=(i+1)*(_M+1)+j-1;
                    count++;
                }
                //case 3.2
                    weights[count]=(slow[i_p][j_m]+slow[i_p][j_p])/2.0;
                    source[count]=(i+1)*(_M+1)+j;
                    count++;
                //case 3.3
                if(j<_M){
                    weights[count]= ROOT_TWO * slow[i][j];
                    source[count]=(i+1)*(_M+1)+j+1;
                    count++;
                }
            }
        }
    }
    destination_offset[nodes]=count;
	//Converting Adjacency Matrix in input to required input for nvgraph


    int n = nodes, nnz = edges*2,    vertex_numsets = 1,    edge_numsets = 1;
    float *weights_h=new float[nnz];
    int *destination_offsets_h=new int[n+1];
    int *source_indices_h=new int[nnz];

    for(int i=0;i<nnz;i++){
    	weights_h[i]=weights[i];
    	source_indices_h[i]=source[i];
    }
    for(int i=0;i<nodes+1;i++){
    destination_offsets_h[i]=destination_offset[i];
    }
    delete weights;
    delete destination_offset;
    delete source;
    //Converting our variables to variables for nvgraph

    float * sssp_1_h;
    void * * vertex_dim; // nvgraph variables _h for host data.
    nvgraphStatus_t status;
    nvgraphHandle_t handle;
    nvgraphGraphDescr_t graph;
    nvgraphCSCTopology32I_t CSC_input;
    hipDataType edge_dimT = HIP_R_32F;
    hipDataType * vertex_dimT;


    // Init host data
    sssp_1_h = new float[n];
    vertex_dim = new void*[vertex_numsets];
    vertex_dimT=new hipDataType[vertex_numsets];
    CSC_input= (nvgraphCSCTopology32I_t)new nvgraphCSCTopology32I_t;
    vertex_dim[0] = (void * ) sssp_1_h;
    vertex_dimT[0] = HIP_R_32F;


    check(nvgraphCreate( & handle));
    check(nvgraphCreateGraphDescr(handle, & graph));
    CSC_input -> nvertices = n;
    CSC_input -> nedges = nnz;
    CSC_input -> destination_offsets = destination_offsets_h;
    CSC_input -> source_indices = source_indices_h;
    check(nvgraphSetGraphStructure(handle, graph, (void * ) CSC_input, NVGRAPH_CSC_32));
    check(nvgraphAllocateVertexData(handle, graph, vertex_numsets, vertex_dimT));
    check(nvgraphAllocateEdgeData(handle, graph, edge_numsets, & edge_dimT));
    check(nvgraphSetEdgeData(handle, graph, (void * ) weights_h, 0));
    int source_vert = 0;
    check(nvgraphSssp(handle, graph, 0, & source_vert, 0));
    check(nvgraphGetVertexData(handle, graph, (void * ) sssp_1_h, 0));


    float* Actual_seed = new float[(_N+1)*(_M+1)];
    delete weights_h;
    delete destination_offsets_h;
    delete source_indices_h;
    free(vertex_dim);
    delete vertex_dimT;
    delete CSC_input;
    check(nvgraphDestroyGraphDescr(handle, graph));
    check(nvgraphDestroy(handle));
    return sssp_1_h;
}
int main(){
  float ** inp;
  int n=ROW,m=COL;
  inp=new float*[n];
  for(int i=0;i<n;i++){
    inp[i]=new float[m];
    for(int j=0;j<m;j++){
      inp[i][j]=1;
    }
  }
  float tpck[m+1];
  float* out=time_finder(inp,n,m);
  for(int i=0;i<m+1;i++){
    tpck[i]=2;

  }
  double sum=0;
  for(int i=0;i<m+1;i++){
    cout<<out[i]<<" ";
    sum=sum+(tpck[i]-out[i])*(tpck[i]-out[i]);
  }
  cout<<endl<<sum<<endl;
  return 0;
}
