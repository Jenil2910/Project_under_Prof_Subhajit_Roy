#include "hip/hip_runtime.h"

#include <nvgraph.h>
#include <bits/stdc++.h>
#include<hip/hip_runtime.h>
#define ROOT_TWO sqrt(2)
using namespace std;
///////////////////////////////////////////////////
void check(nvgraphStatus_t status) {
        if (status != NVGRAPH_STATUS_SUCCESS) {
                printf("ERROR : %d\n", status);
                exit(0);
        }
}
#define ROW 3
#define COL 3
void time_finder(float* sssp_1_h,float *slow,int _N,int _M,float** actualSeed) {
        int n=(_N+1)*(_M+1), vertex_numsets = 1, edge_numsets = 1, nnz=(_N *( _M +1)+(_N+1)*_M+ _M*_N*2)*2; //nnz=edges*2;
        float *weights_h=new float[nnz];
        int *destination_offsets_h=new int[n+1];
        int *source_indices_h=new int[nnz];
        int count=0;
        for(int i=0; i<= _N; i++) {
                for(int j=0; j<= _M; j++) {
                        int index=i*(_M+1)+j;
                        destination_offsets_h[index]=count;
                        int i_m,i_p,j_m,j_p;
                        i_m=(i-1>0 ? i-1 : 0);
                        i_p=(i>_N-1 ? _N-1 : i);
                        j_m=(j-1>0 ? j-1 : 0);
                        j_p=(j>_M-1 ? _M-1 : j);
                        //case 1
                        if(i>0) {
                                //case 1.1
                                if(j>0) {
                                        weights_h[count]= ROOT_TWO * slow[ i_m*(_M) + j_m];
                                        source_indices_h[count]=(i_m)*(_M+1)+j_m;
                                        count++;
                                }
                                //case 1.2
                                weights_h[count]=(slow[i_m*(_M+1) + j_m]+slow[i_m*(_M) + j_p])/2.0;
                                source_indices_h[count]=(i_m)*(_M+1)+j;
                                count++;
                                //case 1.3
                                if(j<_M) {
                                        weights_h[count]= ROOT_TWO * slow[(i-1)*(_M) + j];
                                        source_indices_h[count]=(i_m)*(_M+1)+j+1;
                                        count++;
                                }
                        }
                        //case 2
                        //case 2.1
                        if(j>0) {
                                weights_h[count]=(slow[i_m*(_M+1) + j_m]+slow[i_p*(_M) +j_m])/2.0;
                                source_indices_h[count]=(i_p)*(_M+1)+j_m;
                                count++;
                        }
                        //case 2.2
                        if(j<_M) {
                                weights_h[count]=(slow[i_m*(_M+1) + j_p]+slow[i_p*(_M) + j_p])/2.0;
                                source_indices_h[count]=(i)*(_M+1)+j+1;
                                count++;
                        }
                        //case 3
                        if(i<_N) {
                                //case 3.1
                                if(j>0) {
                                        weights_h[count]= ROOT_TWO * slow[i*(_M) + j-1];
                                        source_indices_h[count]=(i+1)*(_M+1)+j-1;
                                        count++;
                                }
                                //case 3.2
                                weights_h[count]=(slow[i_p*(_M+1) + j_m]+slow[i_p*(_M) + j_p])/2.0;
                                source_indices_h[count]=(i+1)*(_M+1)+j;
                                count++;
                                //case 3.3
                                if(j<_M) {
                                        weights_h[count]= ROOT_TWO * slow[i*(_M) + j];
                                        source_indices_h[count]=(i+1)*(_M+1)+j+1;
                                        count++;
                                }
                        }
                }
        }
        destination_offsets_h[n]=count;
        //Converting Adjacency Matrix in input to required input for nvgraph.

        void * * vertex_dim; // nvgraph variables _h for host data.
        nvgraphStatus_t status;
        nvgraphHandle_t handle;
        nvgraphGraphDescr_t graph;
        nvgraphCSCTopology32I_t CSC_input;
        hipDataType edge_dimT = HIP_R_32F;
        hipDataType * vertex_dimT;
	
	/*cout<<"n : "<<n<<endl;
	cout<<"nnz : "<<nnz<<endl;
	cout<<"Weights Source:"<<endl;
	for(int u=0;u<nnz;u++){
		cout<<u<<" "<<source_indices_h[u]<<" "<<weights_h[u]<<endl;
	}
	cout<<"Doff:"<<endl;
        for(int u=0;u<n+1;u++){
                cout<<destination_offsets_h[u]<<" ";
        }cout<<endl;*/

        // Init host data
        /* *weights_h, *destination_offsets_h, *source_indices_h, n, nnz, vertex_numsets , edge_numsets already defined */
        sssp_1_h = new float[n];
        vertex_dim = new void*[vertex_numsets];
        vertex_dimT=new hipDataType[vertex_numsets];
        CSC_input= (nvgraphCSCTopology32I_t) new nvgraphCSCTopology32I_t;
        vertex_dim[0] = (void * ) sssp_1_h;
        vertex_dimT[0] = HIP_R_32F;


        check(nvgraphCreate( &handle));
        check(nvgraphCreateGraphDescr(handle, &graph));
        CSC_input->nvertices = n;
        CSC_input->nedges = nnz;
        CSC_input->destination_offsets = destination_offsets_h;
        CSC_input->source_indices = source_indices_h;
        check(nvgraphSetGraphStructure(handle, graph, (void * ) CSC_input, NVGRAPH_CSC_32));
        check(nvgraphAllocateVertexData(handle, graph, vertex_numsets, vertex_dimT));
        check(nvgraphAllocateEdgeData(handle, graph, edge_numsets, &edge_dimT));
        check(nvgraphSetEdgeData(handle, graph, (void * ) weights_h, 0));
        int source_vert = 0;
        check(nvgraphSssp(handle, graph, 0, &source_vert, 0));
        check(nvgraphGetVertexData(handle, graph, (void * ) sssp_1_h, 0));

		//////////// To be used to print actual path...
	int parent[n];
	parent[0]=0;
	for(int i=0;i<n;i++){
	int j=i;
	int d_start=destination_offsets_h[j];
	int d_end=destination_offsets_h[j+1];
	int min=d_start;
	for(int y=d_start;y<d_end;y++){
		if(sssp_1_h[source_indices_h[min]]+weights_h[min]>sssp_1_h[source_indices_h[y]]+weights_h[y]){
			min=y;
		}
	}
	j=source_indices_h[min];
	parent[i]=j;
	}
	/*cout<<"i parent[i]"<<endl;
	for(int u=0;u<_M+1;u++){
		cout<<u<<" "<<parent[u]<<endl;
	}*/
	for(int i=1;i<_M+1;i++){
	int node=i;
	//printf("Path to vertex %d\n",i);
	while((node!=0)&&(node!=parent[node])){
		//cout<<node<<" <- "<<parent[node]<<endl;
		int box;
		if(node>parent[node]){
			box=node;
		}else{
			box=parent[node];
		}
		//actualSeed[box/_M][box%_M];
		int diff=node-parent[node];
		if(diff==1 || diff==-1 || diff==_M+1 || diff==-(_M+1)){
			actualSeed[i][ _M*(box/_M) + box%_M]+=1;
		}else{
			actualSeed[i][ _M*(box/_M) + box%_M]+=ROOT_TWO;
		}
		node=parent[node];
	}
	}
	//Use DP for finding the exact path.	
        
        delete weights_h;
        delete destination_offsets_h;
        delete source_indices_h;
        delete vertex_dim;
        delete vertex_dimT;
        delete CSC_input;
        check(nvgraphDestroyGraphDescr(handle, graph));
        check(nvgraphDestroy(handle));
}
int main(){
        float * inp;
        int n=ROW,m=COL;
        inp=new float[n*m];
        for(int i=0; i<n; i++) {
                for(int j=0; j<m; j++) {
                        //inp[i*m+j]=1;
                }
        }
	inp[0]=1;
	inp[1]=100;
	inp[2]=4;
	inp[3]=2;
	inp[4]=3;
	inp[5]=101;
	inp[6]=110;
	inp[7]=103;
	inp[8]=105;
	float** actualSeed;
	actualSeed=new float*[m+1];
	for(int i=0;i<m+1;i++){
		actualSeed[i]=new float[m*n];
	}
	for(int i=0;i<m+1;i++){
		for(int j=0;j<m*n;j++){
			actualSeed[i][j]=0;
		}
	}
        //float tpck[m+1];
        float* out;
	out=new float[(n+1)*(m+1)];
	time_finder(out,inp,n,m,actualSeed);
        /*for(int i=0; i<m+1; i++) {
                tpck[i]=i;
        }
        float sum=0;*/
        for(int i=0; i<m+1; i++) {
                cout<<out[i]<<" ";
                //sum=sum+(tpck[i]-out[i])*(tpck[i]-out[i]);
        }
	cout<<endl;
        //cout<<endl<<sum<<endl;
	//cout<<"-------------"<<endl;
	/*for(int i=0;i<m+1;i++){
                for(int j=0;j<m*n;j++){
                        cout<<actualSeed[i][j]<<" ";
                }cout<<endl;
        }cout<<endl;*/
        return 0;
}
