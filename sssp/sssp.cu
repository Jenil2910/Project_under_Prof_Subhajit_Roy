#include "hip/hip_runtime.h"
#include<iostream>
#include<nvgraph.h>
#include<bits/stdc++.h>
#include<stdio.h>

void check(nvgraphStatus_t status) {
  if (status != NVGRAPH_STATUS_SUCCESS) {
    printf("ERROR : %d\n", status);
    exit(0);
  }
}

/*
This code takes Adjacency Matrix as input and finds the exact shortest path from source to every vertex.
Sample input is provided alongside the code.
Sample output obtained is:

Path to vertex 1
1 <- 0
Path to vertex 2
2 <- 1 <- 0
Path to vertex 3
3 <- 1 <- 0
Path to vertex 4
4 <- 3 <- 1 <- 0

*/




int main(int argc, char * * argv) {

  int edges, nodes;
	scanf("%d %d",&nodes,&edges);
	float Matrix[nodes][nodes];
	for(int i=0;i<nodes;i++){
	    for(int j=0;j<nodes;j++){
	        scanf("%f",&Matrix[i][j]);
	    }
	}
	int d_off=0;
	int edge_itr=0;
	float weights[edges];
	int destination_offset[nodes+1];
	int source[edges];
	for(int j=0;j<nodes;j++){
	    destination_offset[j]=d_off;
	    for(int i=0;i<nodes;i++){
	        if(Matrix[i][j]!=-1){
	            source[edge_itr]=i;
	            weights[edge_itr]=Matrix[i][j];
	            edge_itr++;
	            d_off++;
	        }
	    }
	}
	destination_offset[nodes]=edges;
	//Converting Adjacency Matrix in input to required input for nvgraph
  
  
  const size_t n = nodes, nnz = edges,    vertex_numsets = 1,    edge_numsets = 1;
  float weights_h[nnz];
  int destination_offsets_h[nodes+1];
  int source_indices_h[nnz];
  
  for(int i=0;i<nnz;i++){
		weights_h[i]=weights[i];
		source_indices_h[i]=source[i];
  }
  for(int i=0;i<nodes+1;i++){
	destination_offsets_h[i]=destination_offset[i];
  }
  
  //Converting our variables to variables for nvgraph
  
  float * sssp_1_h;
  void * * vertex_dim; // nvgraph variables _h for host data.
  nvgraphStatus_t status;
  nvgraphHandle_t handle;
  nvgraphGraphDescr_t graph;
  nvgraphCSCTopology32I_t CSC_input;
  hipDataType edge_dimT = HIP_R_32F;
  hipDataType * vertex_dimT;
  
  
  // Init host data 
  sssp_1_h = (float * ) malloc(n * sizeof(float));
  vertex_dim = (void * * ) malloc(vertex_numsets * sizeof(void * ));
  vertex_dimT = (hipDataType * ) malloc(vertex_numsets * sizeof(hipDataType));
  CSC_input = (nvgraphCSCTopology32I_t) malloc(sizeof(struct nvgraphCSCTopology32I_st));
  vertex_dim[0] = (void * ) sssp_1_h;
  vertex_dimT[0] = HIP_R_32F;
  
  /*float weights_h[],  destination_offsets_h[],   source_indices_h[]  are defined earlier...*/
  
  
  check(nvgraphCreate( & handle));
  check(nvgraphCreateGraphDescr(handle, & graph));
  CSC_input -> nvertices = n;
  CSC_input -> nedges = nnz;
  CSC_input -> destination_offsets = destination_offsets_h;
  CSC_input -> source_indices = source_indices_h;
  check(nvgraphSetGraphStructure(handle, graph, (void * ) CSC_input, NVGRAPH_CSC_32));
  check(nvgraphAllocateVertexData(handle, graph, vertex_numsets, vertex_dimT));
  check(nvgraphAllocateEdgeData(handle, graph, edge_numsets, & edge_dimT));
  check(nvgraphSetEdgeData(handle, graph, (void * ) weights_h, 0));
  int source_vert = 0;
  check(nvgraphSssp(handle, graph, 0, & source_vert, 0));
  check(nvgraphGetVertexData(handle, graph, (void * ) sssp_1_h, 0));
  
  
  //sssp_1_h is the array that contains the shortest distance of every vertex from the source.
  
  
  ///////////////////////////////////////////////
  int parent[n];
  parent[0]=0;
  for(int i=1;i<n;i++){
	int j=i;
	int d_start=destination_offsets_h[j];
	int d_end=destination_offsets_h[j+1];
	int min=d_start;
	for(int y=d_start;y<d_end;y++){
		if(sssp_1_h[source_indices_h[min]]+weights_h[min]>sssp_1_h[source_indices_h[y]]+weights_h[y]){
			min=y;
		}
	}
	j=source_indices_h[min];
	parent[i]=j;
  }
  
  for(int i=1;i<n;i++){
	int node=i;
	printf("Path to vertex %d\n",i);
	while((node!=0)&&(node!=parent[node])){
		printf("%d <- ",node);
		node=parent[node];
	}
	printf("0\n");
  }
  
  //Use DP for finding the exact path.
  //////////////////////////////////////////////
  
  free(sssp_1_h);
  free(vertex_dim);
  free(vertex_dimT);
  free(CSC_input);
  check(nvgraphDestroyGraphDescr(handle, graph));
  check(nvgraphDestroy(handle));
  return 0;
}
