#include "hip/hip_runtime.h"
#include<iostream>
#include<nvgraph.h>
#include<bits/stdc++.h>
#include<stdio.h>
#define _N 1    //Number of rows
#define _M 1    //Number of columns
using namespace std;
///////////////////////////////////////////////////
void check(nvgraphStatus_t status) {
  if (status != NVGRAPH_STATUS_SUCCESS) {
    printf("ERROR : %d\n", status);
    exit(0);
  }
}

/*
This code takes Adjacency Matrix as input and finds the exact shortest path from source to every vertex.
Sample input is provided alongside the code.
Sample output obtained is:

Path to vertex 1
1 <- 0
Path to vertex 2
2 <- 1 <- 0
Path to vertex 3
3 <- 1 <- 0
Path to vertex 4
4 <- 3 <- 1 <- 0

*/




int main(int argc, char * * argv) {
    //destination_offsets_h[i]=destination_offset[i];
    //const size_t n = nodes, nnz = edges,    vertex_numsets = 1,    edge_numsets = 1;
    float** slow = new float*[_N];
    for(int i = 0; i < _N; ++i){
        slow[i] = new float[_M];
    }
    //input matrix is taken
    for(int i=0;i<_N;i++){
        for(int j=0;j<_M;j++){
            cin>>slow[i][j];
        }
    }
    int edges=_N *( _M +1)+(_N+1)*_M+ _M * _N*2;
    int nodes=(_N+1)*(_M+1);
    float* weights=new float[edges*2];
    int* destination_offset=new int[nodes+1];
    int* source=new int[edges*2];
    int count=0;
    for(int i=0;i<= _N;i++){
        for(int j=0;j<= _M;j++){
            int index=i*(_M+1)+j;
            destination_offset[index]=count;
            //case 1
            if(i>0){
                //case 1.1
                if(j>0){
                    weights[count]=slow[i-1][j-1];
                    source[count]=(i-1)*(_M+1)+j-1;
                    count++;
                }
                //case 1.2
                if(j>0&&j<_M){
                    weights[count]=(slow[i-1][j-1]+slow[i-1][j])/2.0;
                    source[count]=(i-1)*(_M+1)+j;
                    count++;
                }else if(j==_M){
                    weights[count]=slow[i-1][j-1];
                    source[count]=(i-1)*(_M+1)+j;
                    count++;
                }else{//j==0
                    weights[count]=slow[i-1][j];
                    source[count]=(i-1)*(_M+1)+j;
                    count++;
                }
                //case 1.3
                if(j<_M){
                    weights[count]=slow[i-1][j];
                    source[count]=(i-1)*(_M+1)+j+1;
                    count++;
                }
            }
            //case 2
            //case 2.1
            if(i>0&&j>0&&i<_N){
                weights[count]=(slow[i-1][j-1]+slow[i][j-1])/2.0;
                source[count]=(i)*(_M+1)+j-1;
                count++;
            }else if(i==0&&j>0){
                weights[count]=slow[i][j-1];
                source[count]=(i)*(_M+1)+j-1;
                count++;
            }else if(i==_N&&j>0){
                weights[count]=slow[i-1][j-1];
                source[count]=(i)*(_M+1)+j-1;
                count++;
            }
            //case 2.2
            if(i>0&&j<_M&&i<_N){
                weights[count]=(slow[i-1][j]+slow[i][j])/2.0;
                source[count]=(i)*(_M+1)+j+1;
                count++;
            }else if(i==0&&j<_M){
                weights[count]=slow[i][j];
                source[count]=(i)*(_M+1)+j+1;
                count++;
            }else if(i==_N&&j<_M){
                weights[count]=slow[i-1][j];
                source[count]=(i)*(_M+1)+j+1;
                count++;
            }
            //case 3
            if(i<_N){
                //case 3.1
                if(j>0){
                    weights[count]=slow[i][j-1];
                    source[count]=(i+1)*(_M+1)+j-1;
                    count++;
                }
                //case 3.2
                if(j>0&&j<_M){
                    weights[count]=(slow[i][j-1]+slow[i][j])/2.0;
                    source[count]=(i+1)*(_M+1)+j;
                    count++;
                }else if(j==_M){
                    weights[count]=slow[i][j-1];
                    source[count]=(i+1)*(_M+1)+j;
                    count++;
                }else{//j==0
                    weights[count]=slow[i][j];
                    source[count]=(i+1)*(_M+1)+j;
                    count++;
                }
                //case 3.3
                if(j<_M){
                    weights[count]=slow[i][j];
                    source[count]=(i+1)*(_M+1)+j+1;
                    count++;
                }
            }
        }
    }
    cout<<count;
    destination_offset[nodes]=count;
	//Converting Adjacency Matrix in input to required input for nvgraph

	
    const size_t n = nodes, nnz = edges,    vertex_numsets = 1,    edge_numsets = 1;
    float weights_h[nnz];
    int destination_offsets_h[nodes+1];
    int source_indices_h[nnz];

    for(int i=0;i<nnz;i++){
    	weights_h[i]=weights[i];
    	source_indices_h[i]=source[i];
    }
    for(int i=0;i<nodes+1;i++){
    destination_offsets_h[i]=destination_offset[i];
    }

    //Converting our variables to variables for nvgraph

    float * sssp_1_h;
    void * * vertex_dim; // nvgraph variables _h for host data.
    nvgraphStatus_t status;
    nvgraphHandle_t handle;
    nvgraphGraphDescr_t graph;
    nvgraphCSCTopology32I_t CSC_input;
    hipDataType edge_dimT = HIP_R_32F;
    hipDataType * vertex_dimT;


    // Init host data
    sssp_1_h = (float * ) malloc(n * sizeof(float));
    vertex_dim = (void * * ) malloc(vertex_numsets * sizeof(void * ));
    vertex_dimT = (hipDataType * ) malloc(vertex_numsets * sizeof(hipDataType));
    CSC_input = (nvgraphCSCTopology32I_t) malloc(sizeof(struct nvgraphCSCTopology32I_st));
    vertex_dim[0] = (void * ) sssp_1_h;
    vertex_dimT[0] = HIP_R_32F;

    /*float weights_h[],  destination_offsets_h[],   source_indices_h[]  are defined earlier...*/


    check(nvgraphCreate( & handle));
    check(nvgraphCreateGraphDescr(handle, & graph));
    CSC_input -> nvertices = n;
    CSC_input -> nedges = nnz;
    CSC_input -> destination_offsets = destination_offsets_h;
    CSC_input -> source_indices = source_indices_h;
    check(nvgraphSetGraphStructure(handle, graph, (void * ) CSC_input, NVGRAPH_CSC_32));
    check(nvgraphAllocateVertexData(handle, graph, vertex_numsets, vertex_dimT));
    check(nvgraphAllocateEdgeData(handle, graph, edge_numsets, & edge_dimT));
    check(nvgraphSetEdgeData(handle, graph, (void * ) weights_h, 0));
    int source_vert = 0;
    check(nvgraphSssp(handle, graph, 0, & source_vert, 0));
    check(nvgraphGetVertexData(handle, graph, (void * ) sssp_1_h, 0));


    //sssp_1_h is the array that contains the shortest distance of every vertex from the source.


    ///////////////////////////////////////////////
    int parent[n];
    parent[0]=0;
    for(int i=1;i<n;i++){
    int j=i;
    int d_start=destination_offsets_h[j];
    int d_end=destination_offsets_h[j+1];
    int min=d_start;
    for(int y=d_start;y<d_end;y++){
    	if(sssp_1_h[source_indices_h[min]]+weights_h[min]>sssp_1_h[source_indices_h[y]]+weights_h[y]){
    		min=y;
    	}
    }
    j=source_indices_h[min];
    parent[i]=j;
    }

    for(int i=1;i<n;i++){
    int node=i;
    printf("Path to vertex %d\n",i);
    while((node!=0)&&(node!=parent[node])){
    	printf("%d <- ",node);
    	node=parent[node];
    }
    printf("0\n");
    }

    //Use DP for finding the exact path.
    //////////////////////////////////////////////

    free(sssp_1_h);
    free(vertex_dim);
    free(vertex_dimT);
    free(CSC_input);
    check(nvgraphDestroyGraphDescr(handle, graph));
    check(nvgraphDestroy(handle));
    return 0;
}
